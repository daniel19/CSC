
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>

__global__ void matMulKernel(float* matA, float* matB, float* matC, int rows, int cols)
{
    dim3 gIdx;
    gIdx.y = blockIdx.y * blockDim.y + threadIdx.y;
    gIdx.x = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0;
    if(gIdx.x < cols && gIdx.y < rows)
    {
        for(int i = 0; i < rows; ++i)
        {
             sum += matA[gIdx.y * cols + i] * matB[i * cols + gIdx.x];
        }
        
        matC[gIdx.y * cols + gIdx.x] = sum;
    }
}

void printMat(float* mat, int rows, int cols)
{
    for(int i = 0; i < rows; ++i)
    {
        for(int j = 0; j < cols; ++j)
        {
            int index = i * cols + j;
            std::cout << mat[index] << " "; 
        }
        std::cout << "\n";
    }
}

int main(int argc, char** argv)
{
    if(argc != 2)
    {
        std::cout << "Usage: " << argv[0] << " <DIM>" << std::endl;
        exit(1);
    }

    int matDim = atoi(argv[1]);
    const int NUM_COLS = matDim;
    const int NUM_ROWS = matDim;

    //allocate host mem for input matrices
    float* matA_h = new float[NUM_ROWS * NUM_COLS];
    float* matB_h = new float[NUM_ROWS * NUM_COLS];

    //fill input matrices
    for(int i = 0; i < NUM_ROWS; ++i)
    {
        for(int j = 0; j < NUM_COLS; ++j)
        {
            int index = i * NUM_COLS + j; 
            matA_h[index] = index;
            
            //scale matrix (factor 2)
            matB_h[index] = (i == j) ? 2 : 0;
        }
    }   

    //allocate dev mem for input matrices
    float* matA_d;
    float* matB_d;
    int matSize = NUM_ROWS * NUM_COLS * sizeof(float);
    hipMalloc(&matA_d, matSize);
    hipMalloc(&matB_d, matSize);

    //copy input matrices to device
    hipMemcpy(matA_d, matA_h, matSize, hipMemcpyHostToDevice); 
    hipMemcpy(matB_d, matB_h, matSize, hipMemcpyHostToDevice);

    //allocate dev mem for output matrix
    float* matC_d;
    hipMalloc(&matC_d, matSize);
    hipMemset(matC_d, 0, matSize);
    
    //determine block and grid size
    dim3 bDim(16, 16);
    dim3 gDim;
    gDim.x = (NUM_ROWS + 16 - 1) / 16; //ceil(num_rows/16)
    gDim.y = (NUM_ROWS + 16 - 1) / 16;

    hipEvent_t start, stop;

    //record start event
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //launch kernel
    matMulKernel<<<gDim, bDim>>>(matA_d, matB_d, matC_d, NUM_ROWS, NUM_COLS);
    
    //record stop event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);

    //allocate host mem for output matrix
    float* matC_h = new float[NUM_ROWS * NUM_COLS];

    //copy output matrix from dev to host
    hipMemcpy(matC_h, matC_d, matSize, hipMemcpyDeviceToHost);

    //print output matrix
    printMat(matC_h, NUM_ROWS, NUM_COLS); 

    std::cout << std::endl << "Compute time: " << elapsed << "ms" << std::endl;
}
