#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <stdlib.h>
#include <iomanip>
void  printGrid(float* grid, int rows, int cols);
/**
 *Kernel will update the matrix to keep the heater cells constant.
 */
__global__ void copyHeaters(float* stateGrid, float* heaterGrid, int nRows, int nCols, int iteration) {
    for(int i = 0; i < nRows; i++){
        for(int j = 0; j < nCols; j++){
            float heatValue = heaterGrid[i*nCols +j];
            if(heatValue != 0)
                stateGrid[i*nCols + j] =heatValue;
        }
    }
    printf("\n FOR ITERATION %d \n\n", iteration);
    printGrid(stateGrid, nRows, nCols);
}

__global__ void updateGrid(float* inGrid, float* outGrid, float k, int nRows, int nCols) {
    //Find these values from the inGrid
    int Tlft, Trite, Tup, Tdown;
    for(int i=0; i < nRows; ++i){
       for(int j=0; j < nCols; ++j){
           int currentPosition = i*nCols+j;
           Tlft = currentPosition +1;
           Trite = currentPosition -1;
           Tup = currentPosition -nCols;
           Tdown = currentPosition +nCols;
           float Tnew = inGrid[currentPosition];
           float Top, Tbottom, Tleft, Tright;
           Tbottom = (Tdown < 0) ? 0 : inGrid[Tdown];
           Top = (Tup > nCols) ? 0 : inGrid[Tup];
           Tright = (Trite < 0) ?  0 : inGrid[Trite];
           Tleft = (Tlft > nCols) ? 0 : inGrid[Tlft]; 
           
            Tnew = Tnew + k*(Top + Tbottom + Tleft + Tright - (4*Tnew));

           outGrid[currentPosition] = Tnew;
       }
   }
}

/*------------------------------------------------------------------------------
readHeaterFile

Assumes heaterGrid points to a flattened 2d array of size [rows,cols]
Fille heaterGrid with heaters from the heater file
------------------------------------------------------------------------------*/
void readHeaterFile(const char* fileName, float* heaterGrid, int rows, int cols) {
    std::ifstream inFile(fileName); 

    int numHeaters;
    inFile >> numHeaters;

    for(int i = 0; i < numHeaters; ++i) {
        int hRow, hCol;
        inFile >> hRow;
        inFile >> hCol;
        
        float temp;
        inFile >> temp; 

        heaterGrid[hRow * cols + hCol] = temp;
    }

    inFile.close();
}


/*------------------------------------------------------------------------------
printGrid
------------------------------------------------------------------------------*/
__device__ void printGrid(float* grid, int rows, int cols) {
    //std::cout << std::fixed << std::setprecision(2);
    for(int i = 0; i < rows; ++i) {
        for(int j = 0; j < cols; ++j) {
     //       std::cout << std::setw(6) << grid[i*cols+j] << " ";
            printf("%f ", grid[i*cols+j]);
        }
        //std::cout << std::endl;
        printf("\n");
    }
}

/*------------------------------------------------------------------------------
printGridToFile
------------------------------------------------------------------------------*/
void printGridToFile(float* grid, int rows, int cols, char* fileName) {
    std::ofstream outFile(fileName);
    outFile << std::fixed << std::setprecision(2);
    for(int i = 0; i < rows; ++i) {
        for(int j = 0; j < cols; ++j) {
            outFile << std::setw(6) << grid[i*cols+j] << " ";
        }
        outFile<< std::endl;
    }
    outFile.close();
}

/*------------------------------------------------------------------------------
main
------------------------------------------------------------------------------*/
int main(int argc, char** argv) {
    if(argc != 6) {
        std::cout << "Usage: " << argv[0] << 
            " <numRows> <numCols> <k> <timesteps> <heaterFileName>" << std::endl;

        return 0;
    }

    //Input arguments
    int rows = atoi(argv[1]);
    int cols = atoi(argv[2]);
    float k = atof(argv[3]);
    int timeSteps = atoi(argv[4]);

    //Allocate heater grid
    int gridSize = rows * cols * sizeof(float);
    float* heaterGrid_h = (float*)malloc(gridSize);

    //Read in heater file
    readHeaterFile(argv[5], heaterGrid_h, rows, cols); 

    float* heaterGrid_d; //device pointer
    //TODO Copy heater grid to device
    hipMalloc(&heaterGrid_d, gridSize);
    hipMemcpy(heaterGrid_d, heaterGrid_h, gridSize, hipMemcpyHostToDevice);


    //Input grid
    float* inGrid_h = (float*)malloc(gridSize);
    memset(inGrid_h, 0, gridSize);
    float* inGrid_d; //device pointer
    //TODO Allocate and copy inGrid to device
    hipMalloc(&inGrid_d, gridSize);
    hipMemcpy(inGrid_d, inGrid_h, gridSize, hipMemcpyHostToDevice);

    //Output grid
    float* outGrid_h = (float*)malloc(gridSize);
    memset(outGrid_h, 0, gridSize);
    float* outGrid_d; //device pointer
    //TODO Allocate and copy outGrid to device
    hipMalloc(&outGrid_d, gridSize);
    hipMemcpy(outGrid_d, outGrid_h, gridSize, hipMemcpyHostToDevice);

    //TODO fill in update loop
    for(int i = 0; i < timeSteps; ++i) {
        //copy heater temps to inGrid_d (kernel call)
        copyHeaters<<<1, 1>>>(inGrid_d, heaterGrid_d, rows, cols, i);
        //update outGrid_d based on inGrid_d (kernel call)
        updateGrid<<<1, 1>>>(inGrid_d, outGrid_d, k, rows, cols);
        //swap pointers inGrid_d and outGrid_d
        float* temp = inGrid_d;
        inGrid_d = outGrid_d;
        outGrid_d = temp;
    }
    
    //TODO copy inGrid_d back to host (to inGrid_h)
    hipMemcpy(inGrid_h, inGrid_d, gridSize, hipMemcpyDeviceToHost); 
    printGridToFile(inGrid_h, rows, cols, "output.txt");

    return 0;
}
