
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <stdlib.h>
#include <iomanip>

/**
 *Kernel will update the matrix to keep the heater cells constant.
 */
__global__ void copyHeaters(float* stateGrid, float* heaterGrid, int nRows, int nCols) {
    for(int i = 0; i < nRows; i++){
        for(int j = 0; j < nCols; j++){
            float heatValue = heaterGrid[i*nCols +j];
            if(heatValue != 0)
                stateGrid[i*nCols + j] =heatValue;
        }
    }
}

__global__ void updateGrid(float* inGrid, float* outGrid, float k, int nRows, int nCols) {
    //Find these values from the inGrid
    int Tlft, Trite, Tup, Tdown;
    for(int i=0; i < nRows; ++i){
       for(int j=0; j < nCols; ++j){
           int currentPosition = i*nCols+j;
           Tlft = currentPosition +1;
           Trite = currentPosition -1;
           Tup = currentPosition +4;
           Tdown = currentPosition -4;
           float Tnew = inGrid[currentPosition];
           float Top, Tbottom, Tleft, Tright;
           Tbottom = (Tdown < 0) ? 0 : inGrid[Tdown];
           Top = (Tup > nCols) ? 0 : inGrid[Tup];
           Tright = (Trite < 0) ?  0 : inGrid[Trite];
           Tleft = (Tlft > nCols) ? 0 : inGrid[Tlft]; 
           
            Tnew = Tnew + k*(Top + Tbottom + Tleft + Tright - (4*Tnew));

           outGrid[currentPosition] = Tnew;
       }
   } 
}

/*------------------------------------------------------------------------------
readHeaterFile

Assumes heaterGrid points to a flattened 2d array of size [rows,cols]
Fille heaterGrid with heaters from the heater file
------------------------------------------------------------------------------*/
void readHeaterFile(const char* fileName, float* heaterGrid, int rows, int cols) {
    std::ifstream inFile(fileName); 

    int numHeaters;
    inFile >> numHeaters;

    for(int i = 0; i < numHeaters; ++i) {
        int hRow, hCol;
        inFile >> hRow;
        inFile >> hCol;
        
        float temp;
        inFile >> temp; 

        heaterGrid[hRow * cols + hCol] = temp;
    }

    inFile.close();
}


/*------------------------------------------------------------------------------
printGrid
------------------------------------------------------------------------------*/
void printGrid(float* grid, int rows, int cols) {
    std::cout << std::fixed << std::setprecision(2);
    for(int i = 0; i < rows; ++i) {
        for(int j = 0; j < cols; ++j) {
            std::cout << std::setw(6) << grid[i*cols+j] << " ";
        }
        std::cout << std::endl;
    }
}

/*------------------------------------------------------------------------------
printGridToFile
------------------------------------------------------------------------------*/
void printGridToFile(float* grid, int rows, int cols, char* fileName) {
    std::ofstream outFile(fileName);
    outFile << std::fixed << std::setprecision(2);
    for(int i = 0; i < rows; ++i) {
        for(int j = 0; j < cols; ++j) {
            outFile << std::setw(6) << grid[i*cols+j] << " ";
        }
        outFile<< std::endl;
    }
    outFile.close();
}

/*------------------------------------------------------------------------------
main
------------------------------------------------------------------------------*/
int main(int argc, char** argv) {
    if(argc != 6) {
        std::cout << "Usage: " << argv[0] << 
            " <numRows> <numCols> <k> <timesteps> <heaterFileName>" << std::endl;

        return 0;
    }

    //Input arguments
    int rows = atoi(argv[1]);
    int cols = atoi(argv[2]);
    float k = atof(argv[3]);
    int timeSteps = atoi(argv[4]);

    //Allocate heater grid
    int gridSize = rows * cols * sizeof(float);
    float* heaterGrid_h = (float*)malloc(gridSize);

    //Read in heater file
    readHeaterFile(argv[5], heaterGrid_h, rows, cols); 

    float* heaterGrid_d; //device pointer
    //TODO Copy heater grid to device
    hipMalloc(&heaterGrid_d, gridSize);
    hipMemcpy(heaterGrid_d, heaterGrid_h, gridSize, hipMemcpyDeviceToHost);


    //Input grid
    float* inGrid_h = (float*)malloc(gridSize);
    memset(inGrid_h, 0, gridSize);
    float* inGrid_d; //device pointer
    //TODO Allocate and copy inGrid to device
    hipMalloc(&inGrid_d, gridSize);
    hipMemcpy(inGrid_d, inGrid_h, gridSize, hipMemcpyDeviceToHost);

    //Output grid
    float* outGrid_h = (float*)malloc(gridSize);
    memset(outGrid_h, 0, gridSize);
    float* outGrid_d; //device pointer
    //TODO Allocate and copy outGrid to device
    hipMalloc(&outGrid_d, gridSize);
    hipMemcpy(outGrid_d, outGrid_h, gridSize, hipMemcpyDeviceToHost);

    //TODO fill in update loop
    for(int i = 0; i < timeSteps; ++i) {
        //copy heater temps to inGrid_d (kernel call)
        copyHeaters<<<1, 1>>>(inGrid_d, heaterGrid_d, rows, cols);
        //update outGrid_d based on inGrid_d (kernel call)
        //swap pointers inGrid_d and outGrid_d
    }
    
    //TODO copy inGrid_d back to host (to inGrid_h)
    hipMemcpy(inGrid_h, inGrid_d, gridSize, hipMemcpyHostToDevice); 
    printGridToFile(inGrid_h, rows, cols, "output.txt");

    return 0;
}
