
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <stdlib.h>
#include <iomanip>

__global__ void copyHeaters(float* stateGrid, float* heaterGrid, int nRows, int nCols) {
   //TODO 
}

__global__ void updateGrid(float* inGrid, float* outGrid, float k, int nRows, int nCols) {
   //TODO 
}

/*------------------------------------------------------------------------------
readHeaterFile

Assumes heaterGrid points to a flattened 2d array of size [rows,cols]
Fille heaterGrid with heaters from the heater file
------------------------------------------------------------------------------*/
void readHeaterFile(const char* fileName, float* heaterGrid, int rows, int cols) {
    std::ifstream inFile(fileName); 

    int numHeaters;
    inFile >> numHeaters;

    for(int i = 0; i < numHeaters; ++i) {
        int hRow, hCol;
        inFile >> hRow;
        inFile >> hCol;
        
        float temp;
        inFile >> temp; 

        heaterGrid[hRow * cols + hCol] = temp;
    }

    inFile.close();
}


/*------------------------------------------------------------------------------
printGrid
------------------------------------------------------------------------------*/
void printGrid(float* grid, int rows, int cols) {
    std::cout << std::fixed << std::setprecision(2);
    for(int i = 0; i < rows; ++i) {
        for(int j = 0; j < cols; ++j) {
            std::cout << std::setw(6) << grid[i*cols+j] << " ";
        }
        std::cout << std::endl;
    }
}

/*------------------------------------------------------------------------------
printGridToFile
------------------------------------------------------------------------------*/
void printGridToFile(float* grid, int rows, int cols, char* fileName) {
    std::ofstream outFile(fileName);
    outFile << std::fixed << std::setprecision(2);
    for(int i = 0; i < rows; ++i) {
        for(int j = 0; j < cols; ++j) {
            outFile << std::setw(6) << grid[i*cols+j] << " ";
        }
        outFile<< std::endl;
    }
    outFile.close();
}

/*------------------------------------------------------------------------------
main
------------------------------------------------------------------------------*/
int main(int argc, char** argv) {
    if(argc != 6) {
        std::cout << "Usage: " << argv[0] << 
            " <numRows> <numCols> <k> <timesteps> <heaterFileName>" << std::endl;

        return 0;
    }

    //Input arguments
    int rows = atoi(argv[1]);
    int cols = atoi(argv[2]);
    float k = atof(argv[3]);
    int timeSteps = atoi(argv[4]);

    //Allocate heater grid
    int gridSize = rows * cols * sizeof(float);
    float* heaterGrid_h = (float*)malloc(gridSize);

    //Read in heater file
    readHeaterFile(argv[5], heaterGrid_h, rows, cols); 

    float* heaterGrid_d; //device pointer
    //TODO Copy heater grid to device

    //Input grid
    float* inGrid_h = (float*)malloc(gridSize);
    memset(inGrid_h, 0, gridSize);
    float* inGrid_d; //device pointer
    //TODO Allocate and copy inGrid to device

    //Output grid
    float* outGrid_h = (float*)malloc(gridSize);
    memset(outGrid_h, 0, gridSize);
    float* outGrid_d; //device pointer
    //TODO Allocate and copy outGrid to device

    //TODO fill in update loop
    for(int i = 0; i < timeSteps; ++i) {
        //copy heater temps to inGrid_d (kernel call)
        //update outGrid_d based on inGrid_d (kernel call)
        //swap pointers inGrid_d and outGrid_d
    }
    
    //TODO copy inGrid_d back to host (to inGrid_h)
    
    printGridToFile(inGrid_h, rows, cols, "output.txt");

    return 0;
}
