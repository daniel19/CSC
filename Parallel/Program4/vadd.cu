/*
	A simple CUDA test program that adds two vectors
*/


#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vAdd(int* a, int* b, int* c, int N)
{
	int gIdx = (blockDim.x * blockIdx.x) + threadIdx.x;
	
	if(gIdx < N)
	{
		c[gIdx] = a[gIdx] + b[gIdx];
	}
}

int main(int argc, char** argv)
{
    if(argc != 2) {
        std::cout << "Usage: " << argv[0] << " <SIZE>" << std::endl;
        exit(1);
    }

	const int N = atoi(argv[1]);;

	//Allocate on host
	int* h_a = new int[N];
	int* h_b = new int[N];
	int* h_c = new int[N];
	
	//Allocate on device
	int* d_a = NULL;
	int* d_b = NULL;
	int* d_c = NULL;
	hipMalloc(&d_a, sizeof(int) * N);
	hipMalloc(&d_b, sizeof(int) * N);
	hipMalloc(&d_c, sizeof(int) * N);
	
	//Fill input vectors
	for(int i = 0; i < N; ++i)
	{
		h_a[i] = i;
		h_b[i] = i;
		h_c[i] = 0;
	}
	
	//Copy vectors to device
	hipMemcpy(d_a, h_a, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(int) * N, hipMemcpyHostToDevice);
	
	//Call kernel
	int threadsPerBlock = 256;
	int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock; //ceil(n/threadsPerBlock)

    hipEvent_t start, stop;

    //record start event
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

	vAdd<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, N);
    
    //record stop event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
	
	//Copy output vector back to host
	hipMemcpy(h_c, d_c, sizeof(int) * N, hipMemcpyDeviceToHost);
	
	//Print the output vector
    for(int i = 0; i < N; ++i)
    {
        std::cout << h_c[i] << " ";

        if((i + 1) % 15 == 0)
            std::cout << std::endl;
    }

    std::cout << std::endl << "Compute time: " << elapsed << "ms" << std::endl;
}
